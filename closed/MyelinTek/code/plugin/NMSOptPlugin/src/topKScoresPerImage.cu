#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hipcub/hipcub.hpp>
#include <vector>

#include "ssdOpt.h"
#include "ssdOptMacros.h"

template <typename KeyT, typename ValueT>
size_t cubSortPairsWorkspaceSize(int num_items, int num_segments)
{
    size_t temp_storage_bytes = 0;
    hipcub::DeviceSegmentedRadixSort::SortPairsDescending(
        (void*) NULL, temp_storage_bytes,
        (const KeyT*) NULL, (KeyT*) NULL,
        (const ValueT*) NULL, (ValueT*) NULL,
        num_items,    // # items
        num_segments, // # segments
        (const int*) NULL, (const int*) NULL);
    return temp_storage_bytes;
}


namespace nvinfer1
{
namespace plugin
{

namespace {
// sort one segment per cta
template<typename T_SCORE, int BLOCK_THREADS, int ELEMENTS_PER_THREAD>
__global__ void blockSortKernel(const T_SCORE *d_keys_in, T_SCORE *d_keys_out, const int *d_values_in, int *d_values_out, int* active_count_per_batch, int num_items, int stride_items, int num_segments)
{
    // Specialize BlockRadixSort for a 1D block
    typedef hipcub::BlockRadixSort<T_SCORE, BLOCK_THREADS, ELEMENTS_PER_THREAD, int> BlockRadixSort;

    // Allocate shared memory for BlockRadixSort
    __shared__ typename BlockRadixSort::TempStorage temp_storage;

    if (blockIdx.x >= num_segments)
        return;

    int num_active_items = active_count_per_batch[blockIdx.x];

    // Obtain a segment of consecutive items that are blocked across threads
    T_SCORE thread_keys[ELEMENTS_PER_THREAD];
    int thread_values[ELEMENTS_PER_THREAD];

    int block_offset = blockIdx.x * stride_items;
    hipcub::LoadDirectStriped<BLOCK_THREADS>(threadIdx.x, d_keys_in + block_offset, thread_keys, num_active_items, 0);
    hipcub::LoadDirectStriped<BLOCK_THREADS>(threadIdx.x, d_values_in + block_offset, thread_values, num_active_items, -1);
    __syncthreads();

    // Collectively sort the keys and values among block threads
    BlockRadixSort(temp_storage).SortDescendingBlockedToStriped(thread_keys, thread_values);

    // Store output in striped fashion
    hipcub::StoreDirectStriped<BLOCK_THREADS>(threadIdx.x, d_keys_out + block_offset, thread_keys, num_items);
    hipcub::StoreDirectStriped<BLOCK_THREADS>(threadIdx.x, d_values_out + block_offset, thread_values, num_items);
}

/// block sort kernel
template <typename T_SCORE>
void blockSort(const T_SCORE *d_keys_in, T_SCORE *d_keys_out, const int *d_values_in, int *d_values_out, int* active_count_per_batch, int num_items, int stride_items, int num_segments, hipStream_t stream)
{
    if (num_items == 0)
        return;

    int warps_per_cta = (num_items + 31) / 32;
    assert(warps_per_cta <= 8);

    dim3 block(warps_per_cta * 32);
    dim3 grid(num_segments);

    using kernel_func = void (*)(const T_SCORE *d_keys_in, T_SCORE *d_keys_out, const int *d_values_in, int *d_values_out, int* active_count_per_batch, int num_items, int stride_items, int num_segments);

    static const kernel_func kernel_funcs[] = {
        &blockSortKernel<T_SCORE, 32, 1>,
        &blockSortKernel<T_SCORE, 64, 1>,
        &blockSortKernel<T_SCORE, 96, 1>,
        &blockSortKernel<T_SCORE, 128, 1>,
        &blockSortKernel<T_SCORE, 160, 1>,
        &blockSortKernel<T_SCORE, 192, 1>,
        &blockSortKernel<T_SCORE, 224, 1>,
        &blockSortKernel<T_SCORE, 256, 1>,
    };
    kernel_funcs[warps_per_cta - 1]<<<grid, block, 0, stream>>>(d_keys_in, d_keys_out, d_values_in, d_values_out, active_count_per_batch, num_items, stride_items, num_segments);
}

static __host__ __device__ inline int div_up(int m, int n) {
  return (m + n - 1) / n;
}

//#undef SSD_STABLE_TOPK
struct BlockPrefixCallbackOp
{
    // Running prefix
    int running_total;
    // Constructor
    __device__ BlockPrefixCallbackOp(int running_total) : running_total(running_total) {}
    // Callback operator to be entered by the first warp of threads in the block.
    // Thread-0 is responsible for returning a value for seeding the block-wide scan.
    __device__ int operator()(int block_aggregate)
    {
        int old_prefix = running_total;
        running_total += block_aggregate;
        return old_prefix;
    }
};

template <int BLOCK_THREADS> 
__global__ void segmented_scan(int *in, int *out, int *aggregate, int* max, int segments){
  // Specialize BlockScan type for our thread block
  // can be in-place
  typedef hipcub::BlockScan<int, BLOCK_THREADS> BlockScan;
  __shared__ typename BlockScan::TempStorage temp_storage;
  // Initialize running total
  BlockPrefixCallbackOp prefix_op(0);

  const int offset = blockIdx.x * segments;
  in += offset;
  out += offset;

  int finish = div_up(segments, BLOCK_THREADS) * BLOCK_THREADS;
  for (int idx = threadIdx.x; idx < finish; idx += BLOCK_THREADS) {
    int thread_count = (idx < segments)? in[idx] : 0;

    int thread_offset;
    BlockScan(temp_storage).ExclusiveSum(thread_count, thread_offset, prefix_op);
    
    __syncthreads();

    if (idx < segments) out[idx] = thread_offset;
  }
  
  if (threadIdx.x == 0) {
    aggregate[blockIdx.x] = prefix_op.running_total;
    // TODO: we can detect num_items based on the outcome of reduction
/*     if (max != NULL) {
      atomicMax(max, prefix_op.running_total);
    } */
  }
}

__global__ void compact_segments(const int *in, const int * in_indices, 
                                int *out, int* out_indices, 
                                int* active_count_scan, int* active_count_per_batch,
                                int items, int stride) {
// Each batch contain "segment" pieces
// The kernel compacts segments into the beginning of each batch 
// items = max number of items per batch
// items % segment == 0
  //  number of segments per batch
  const int num_segment = gridDim.x;
  const int batch_id = blockIdx.y;

  active_count_scan += batch_id * num_segment;

  const int segment_id = blockIdx.x;
  int segment_size = items / num_segment;
  
  int offset_in = batch_id * stride + segment_id * segment_size;
  int offset_out = batch_id * stride + active_count_scan[segment_id];

  int items_per_segment = (segment_id < num_segment - 1) ?  
                              active_count_scan[segment_id + 1] - active_count_scan[segment_id] 
                            : active_count_per_batch[batch_id] - active_count_scan[segment_id];

  for (int idx = threadIdx.x; idx < items_per_segment; idx += blockDim.x) {
    out[offset_out + idx] = in[offset_in + idx];
    out_indices[offset_out + idx] = in_indices[offset_in + idx];
  }
}

template <int ITEMS_PER_THREAD, int BLOCK_THREADS>
__global__ void top_k_cuda(int *in, int *in_indices, int *out, int* out_indices, 
                          int* active_count, int* active_count_per_batch, 
                          int items, int stride, unsigned int num_top_k)
{
  extern __shared__ uint32_t dynamic_memory[];
  uint32_t* selected_items = dynamic_memory;
  int32_t* selected_indices = reinterpret_cast<int32_t*>(selected_items + num_top_k);
  __shared__ unsigned int selected_count;
  unsigned int old_selected_count;

  // Specialize BlockScan type for our thread block
  #ifdef SSD_STABLE_TOPK
  typedef hipcub::BlockScan<int, BLOCK_THREADS> BlockScan;
  __shared__ typename BlockScan::TempStorage temp_storage;
  // Initialize running total
  BlockPrefixCallbackOp prefix_op(0);
  #endif

  int batch = blockIdx.x;
  int first_index = batch * stride;

  // segments per batch
  int num_segments = gridDim.y;
  int items_per_segment = div_up(items, num_segments);
  int segment_items_offset = blockIdx.y * items_per_segment;
  items = active_count_per_batch[batch];
  if (items < segment_items_offset) {
    //active_count[blockIdx.x * num_segments + blockIdx.y] = 0;
    //return;
    items = 0;
  }

  items = min(items_per_segment, items - segment_items_offset);
  int second_index = first_index + blockIdx.y * num_top_k;
  first_index += segment_items_offset;

  in += first_index;
  in_indices += first_index;

  out += second_index;
  out_indices += second_index;

  // Feed input
  uint32_t thread_items[ITEMS_PER_THREAD];
  int32_t thread_indices[ITEMS_PER_THREAD];

  for (int i = 0; i < ITEMS_PER_THREAD; ++i) {
    int offset = threadIdx.x + i * blockDim.x;
    if (offset < items) {
      thread_items[i] = in[offset];
      thread_indices[i] = in_indices[offset];
    }
     else {
      thread_items[i] = 0;
      thread_indices[i] = -1;
    }
  }

  if (items <= num_top_k) {
      if (threadIdx.x == 0) {
          if (gridDim.y == 1) {
            active_count_per_batch[batch] = items;
          } else {
            // this is preliminary step, so we need to populate active_count
            active_count[blockIdx.x * num_segments + blockIdx.y] = items;
          }
      }

      // we know that the results are compact, so we can bail out early.
      for (int i = 0; i < ITEMS_PER_THREAD; ++i) {
          int offset = threadIdx.x + i * blockDim.x;
            if (offset < num_top_k) {
              out[offset] = thread_items[i];
              out_indices[offset] = thread_indices[i];
          }
          else {
            return;
          }
      }
  }

  uint32_t select_mask = 0;
  uint32_t save_mask = 0;
  uint32_t save_bit = 0;

  if (threadIdx.x == 0) {
    selected_count = 0;
    old_selected_count = 0;
  }

  #define MTA_D 0

  // iterate over bits.
  // skip the first two bits,
  // * bit 31 is the sign bit. all values are positive
  // * bit 30 is only set for values >= 2, but the input consists only of values in the range of [0,1]
  const int skip_bits = 0;
  int selected = 0;
  for (int bit = 31 - skip_bits; true; --bit) {
    __syncthreads();
    uint32_t bit_mask = select_mask | (1u << bit);

    uint32_t enabled = 0;
    for (int item = 0; item < ITEMS_PER_THREAD; ++item) {
        enabled |= (((thread_items[item] ^ bit_mask) & bit_mask) == 0) << item;
    }

    selected = __popc(enabled);
#ifdef SSD_STABLE_TOPK
    int offset;
    BlockScan(temp_storage).ExclusiveSum(selected, offset, prefix_op);
    if (threadIdx.x == 0) {
        selected_count = prefix_op.running_total;
    }
#else
    unsigned int offset = atomicAdd(&selected_count,selected);
#endif

    __syncthreads();
    int sc = selected_count;
    __syncthreads();

    if ((sc <= num_top_k && sc > 0) || (bit == 0 && sc > 0)) {
      for (int item = 0; item < ITEMS_PER_THREAD; ++item) {
         if (enabled & (1u << item) && offset < num_top_k) {
           selected_items[offset] = thread_items[item];
           selected_indices[offset] = thread_indices[item];
           ++offset;
           thread_items[item] = 0;
         }
       }

    }

    if (sc == num_top_k || bit == 0) {
        break;
    }
    else if (sc > num_top_k)
    {
        // There are too many bits in the current selection
        // Save the current state and go to the next bit
        // If there are not enough items left using the next bit
        // it's necessary to restart here with the current bit not set
        save_mask = bit_mask;
        save_bit = bit - 1;
        select_mask |= bit_mask;

        if (threadIdx.x == 0)
        {
            selected_count = old_selected_count;
#ifdef SSD_STABLE_TOPK
            prefix_op.running_total = old_selected_count;
#endif
        }
    }
    else {
        if (save_mask) {
            select_mask = save_mask;
            bit = save_bit;

            save_mask = 0;
        }
        if (threadIdx.x == 0) {
            old_selected_count = sc;
        }
    }
  }

  __syncthreads();

  // store data to global memory
  int sc = selected_count;
  for (int i = threadIdx.x; i < num_top_k; i += blockDim.x) {
      out[i] = (i < sc) ? selected_items[i] : 1;
      out_indices[i] = (i < sc && selected_items[0] > 0) ? selected_indices[i] : -1;
  }

  if ( threadIdx.x == 0) {
    if (gridDim.y == 1) {
      active_count_per_batch[batch] = num_top_k;
    } else {
      active_count[batch*num_segments + blockIdx.y] = num_top_k;
    }
  }

}

}

using top_k_kernel = void (*)(int *in, int *in_indices, int *out, int* out_indices, 
    int* active_count, int* active_count_per_batch, 
    int items, int stride_items, unsigned int num_top_k);

template <int BLOCK_THREADS>
void top_k(top_k_kernel* kernel, int kernel_index, int* unsorted_scores, int*unsorted_bbox_indices, 
           int* sorted_scores, int*sorted_bbox_indices, int* active_count, int* active_count_per_batch,
           int num_items_per_image, int image_stride, int num_top_k, int num_classes, int num_images, 
           int num_segments, hipStream_t& stream) {

  assert(num_items_per_image % num_classes == 0);

  const int scan_block_sz = 128;

  // recomputes active_count_per_batch
  segmented_scan<scan_block_sz><<<num_images, scan_block_sz, 0, stream>>>
                                  (active_count, active_count, active_count_per_batch, NULL, num_classes); 

  compact_segments<<<dim3(num_classes, num_images, 1), 128, 0, stream>>>
                      ( (int*)unsorted_scores, (int*)unsorted_bbox_indices,
                      (int*)sorted_scores, (int*)sorted_bbox_indices,
                      (int*)active_count, (int*)active_count_per_batch,
                      num_items_per_image, image_stride );

  uint32_t smem_size = num_top_k * (sizeof(int) + sizeof(uint32_t));
  
  kernel[kernel_index]<<<dim3(num_images, num_segments, 1), BLOCK_THREADS, smem_size, stream>>>((int*) 
                        (sorted_scores), (int*)sorted_bbox_indices, 
                        (int*) (unsorted_scores), (int*)unsorted_bbox_indices, 
                        (int*)active_count, (int*)active_count_per_batch, 
                        num_items_per_image, image_stride, num_top_k);
}

template <int BLOCK_THREADS>
void top_k_multi_stage(top_k_kernel* top_k_kernels, int* unsorted_scores, int*unsorted_bbox_indices, 
           int* sorted_scores, int*sorted_bbox_indices, int* active_count, int* active_count_per_batch,
           int num_items_per_image, int image_stride, 
           int num_top_k, int num_classes, int num_images, hipStream_t& stream) {
  
  int kernel_index = div_up(num_items_per_image, BLOCK_THREADS);
  
  int num_segments = 1;
  while (kernel_index >= 32) {
      // introduce additional step
      num_segments += 1;
      int items_per_sub_segment = div_up(num_items_per_image, num_segments);
      kernel_index = (items_per_sub_segment + BLOCK_THREADS - 1) / BLOCK_THREADS;
  }

  top_k<BLOCK_THREADS>(top_k_kernels, kernel_index, unsorted_scores, unsorted_bbox_indices, 
                       sorted_scores, sorted_bbox_indices, 
                       active_count, active_count_per_batch, 
                       num_items_per_image, image_stride, num_top_k, num_classes, num_images, num_segments, stream);
  if (num_segments > 1) {
    int num_items_per_image_stage_next = num_segments * num_top_k;
    kernel_index = div_up(num_items_per_image_stage_next, BLOCK_THREADS);
    if (kernel_index >= 32) {
      top_k_multi_stage<BLOCK_THREADS>(top_k_kernels, unsorted_scores, unsorted_bbox_indices, 
                           sorted_scores, sorted_bbox_indices, 
                           active_count, active_count_per_batch, 
                           num_items_per_image_stage_next, image_stride, 
                           num_top_k, num_segments, num_images, stream);
    } else {
      top_k<BLOCK_THREADS>(top_k_kernels, kernel_index, unsorted_scores, unsorted_bbox_indices, 
                           sorted_scores, sorted_bbox_indices, 
                           active_count, active_count_per_batch, 
                           num_items_per_image_stage_next, image_stride, 
                           num_top_k, num_segments, num_images, 1, stream);
    }
  }
}

template <typename T_SCORE>
ssdStatus_t topKScoresPerImage_gpu(
    hipStream_t stream,
    const int num_images,
    const int num_items_per_image,
    const int num_top_k,
    void* unsorted_scores,
    void* unsorted_bbox_indices,
    void* sorted_scores,
    void* sorted_bbox_indices,
    void* active_count,
    void* active_count_per_batch,
    void* workspace)
{
    void* d_offsets = workspace;
    void* cubWorkspace = nextWorkspacePtr((int8_t*) d_offsets, (num_images + 1) * sizeof(int));

    uint32_t num_warps = (num_items_per_image > 1024) ? 32 : (num_items_per_image + 31) / 32;

    const int WARP_SZ = 32;
    const int BLOCK_THREADS = 512;

    const int num_classes = num_items_per_image / num_top_k;

    dim3 block(num_warps * WARP_SZ);
    dim3 grid(num_images);

    top_k_kernel top_k_kernels[] = {
        top_k_cuda<1, BLOCK_THREADS>,
        top_k_cuda<2, BLOCK_THREADS>,
        top_k_cuda<3, BLOCK_THREADS>,
        top_k_cuda<4, BLOCK_THREADS>,
        top_k_cuda<5, BLOCK_THREADS>,
        top_k_cuda<6, BLOCK_THREADS>,
        top_k_cuda<7, BLOCK_THREADS>,
        top_k_cuda<8, BLOCK_THREADS>,
        top_k_cuda<9, BLOCK_THREADS>,
        top_k_cuda<10, BLOCK_THREADS>,
        top_k_cuda<11, BLOCK_THREADS>,
        top_k_cuda<12, BLOCK_THREADS>,
        top_k_cuda<13, BLOCK_THREADS>,
        top_k_cuda<14, BLOCK_THREADS>,
        top_k_cuda<15, BLOCK_THREADS>,
        top_k_cuda<16, BLOCK_THREADS>,
        top_k_cuda<17, BLOCK_THREADS>,
        top_k_cuda<18, BLOCK_THREADS>,
        top_k_cuda<19, BLOCK_THREADS>,
        top_k_cuda<20, BLOCK_THREADS>,
        top_k_cuda<21, BLOCK_THREADS>,
        top_k_cuda<22, BLOCK_THREADS>,
        top_k_cuda<23, BLOCK_THREADS>,
        top_k_cuda<24, BLOCK_THREADS>,
        top_k_cuda<25, BLOCK_THREADS>,
        top_k_cuda<26, BLOCK_THREADS>,
        top_k_cuda<27, BLOCK_THREADS>,
        top_k_cuda<28, BLOCK_THREADS>,
        top_k_cuda<29, BLOCK_THREADS>,
        top_k_cuda<30, BLOCK_THREADS>,
        top_k_cuda<31, BLOCK_THREADS>,
        top_k_cuda<32, BLOCK_THREADS>,
    };

    void * block_sort_scores = NULL;
    void * block_sort_indices = NULL;

#ifdef SSD_STABLE_TOPK
    top_k_multi_stage<BLOCK_THREADS>(top_k_kernels, 
                       (int*) (unsorted_scores), (int*)unsorted_bbox_indices, 
                        (int*) (sorted_scores), (int*)sorted_bbox_indices, 
                        (int*)active_count, (int*)active_count_per_batch, 
                        num_items_per_image, num_items_per_image, 
                        num_top_k, num_classes, num_images, stream);
    block_sort_scores = unsorted_scores;
    block_sort_indices = unsorted_bbox_indices;
#else
    int kernel_index = num_items_per_image / block.x;
    while (kernel_index >= 32) {
        kernel_index /= 2;
        num_warps *= 2;
    }
    assert(kernel_index < 32);
    uint32_t smem_size = num_top_k * (sizeof(int) + sizeof(uint32_t));
    top_k_kernels[kernel_index]<<<grid, BLOCK_THREADS, smem_size, stream>>>((int*) (unsorted_scores), (int*)unsorted_bbox_indices, (int*) (sorted_scores), (int*)sorted_bbox_indices, (int*)active_count, (int*)active_count_per_batch, num_items_per_image, num_items_per_image, num_top_k);
    block_sort_scores = sorted_scores;
    block_sort_indices = sorted_bbox_indices;
#endif

    block.x = num_warps * 32;

    blockSort<T_SCORE>(
                       (const T_SCORE*) (block_sort_scores), (T_SCORE*) (sorted_scores),
                       (const int*) (block_sort_indices), (int*) (sorted_bbox_indices), (int*) active_count_per_batch,
                       num_top_k, num_items_per_image, num_images, stream
    );

    CSC(hipGetLastError(), STATUS_FAILURE);
    return STATUS_SUCCESS;
}

// sortScoresPerImage LAUNCH CONFIG {{{
typedef ssdStatus_t (*tkspiFunc)(hipStream_t,
                                 const int,
                                 const int,
                                 const int,
                                 void*,
                                 void*,
                                 void*,
                                 void*,
                                 void*,
                                 void*,
                                 void*);
struct tkspiLaunchConfig
{
    DType_t t_score;
    tkspiFunc function;

    tkspiLaunchConfig(DType_t t_score)
        : t_score(t_score)
    {
    }
    tkspiLaunchConfig(DType_t t_score, tkspiFunc function)
        : t_score(t_score)
        , function(function)
    {
    }
    bool operator==(const tkspiLaunchConfig& other)
    {
        return t_score == other.t_score;
    }
};

using nvinfer1::DataType;

static std::vector<tkspiLaunchConfig> tkspiFuncVec;
bool tkspiInit()
{
    tkspiFuncVec.push_back(tkspiLaunchConfig(DataType::kFLOAT,
                                           topKScoresPerImage_gpu<float>));
    return true;
}

static bool initialized = tkspiInit();
//}}}

ssdStatus_t topKScoresPerImage(
    hipStream_t stream,
    const int num_images,
    const int num_items_per_image,
    const int num_top_k,
    const DType_t DT_SCORE,
    void* unsorted_scores,
    void* unsorted_bbox_indices,
    void* sorted_scores,
    void* sorted_bbox_indices,
    void* active_count,
    void* active_count_per_gpu,
    void* workspace)
{
    tkspiLaunchConfig lc = tkspiLaunchConfig(DT_SCORE);
    for (unsigned i = 0; i < tkspiFuncVec.size(); ++i)
    {
        if (lc == tkspiFuncVec[i])
        {
            DEBUG_PRINTF("topKScoresPerImage kernel %d\n", i);
            return tkspiFuncVec[i].function(stream,
                                            num_images,
                                            num_items_per_image,
                                            num_top_k,
                                            unsorted_scores,
                                            unsorted_bbox_indices,
                                            sorted_scores,
                                            sorted_bbox_indices,
                                            active_count,
                                            active_count_per_gpu,
                                            workspace);
        }
    }
    return STATUS_BAD_PARAM;
}

size_t topKScoresPerImageWorkspaceSize(
    const int num_images,
    const int num_items_per_image,
    const int num_top_k,
    const DType_t DT_SCORE)
{
    const int arrayLen = num_images * num_items_per_image;
    size_t wss[2];
    wss[0] = (num_images + 1) * sizeof(int); // offsets
    if (DT_SCORE == DataType::kFLOAT)
    {
        wss[1] = cubSortPairsWorkspaceSize<float, int>(arrayLen, num_images); // cub workspace
    }
    else
    {
        printf("SCORE type not supported.\n");
        return (size_t) -1;
    }

    return calculateTotalWorkspaceSize(wss, 2);
}

} // namespace plugin
} // namespace nvinfer1
